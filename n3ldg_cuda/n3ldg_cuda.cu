#include "hip/hip_runtime.h"
#include "n3ldg_cuda.h"
#include <cstdlib>
#include <vector>
#include <algorithm>
#include <cmath>
#include <cstdio>
#include <hipblas.h>
#include "cuPrintf.cuh"
#include "cuPrintf.cu"
#include "memory_pool.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "profiler.h"
#include "cnmem.h"
#include <string>

namespace n3ldg_cuda {

using std::cout;
using std::endl;

#if USE_FLOAT
#define cuda_sqrt(x) sqrtf(x)
#define cuda_pow(x, y) powf(x, y)
#define cuda_tanh(x) tanhf(x)
#else
#define cuda_sqrt(x) sqrt(x)
#define cuda_pow(x, y) pow(x, y)
#define cuda_tanh(x) tanh(x)
#endif

#define KERNEL_LOG

#ifdef KERNEL_LOG
#define  KernelPrintLine(format, ...)\
{\
    cuPrintf("block:x=%d,y=%d thread:x=%d,y=%d "#format"\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y,\
            __VA_ARGS__);\
}
#else
#define KernelPrintLine(format, ...)
#endif

constexpr int THREAD_COUNT_PER_BLOCK = 1024;
constexpr int BLOCK_COUNT = 56;

void CallCuda(hipError_t status) {
    if (status != hipSuccess) {
        cout << hipGetErrorString(status) << endl;
        abort();
    }
}

void CallCnmem(cnmemStatus_t status) {
    assert(status == CNMEM_STATUS_SUCCESS);
}

void CallCublas(hipblasStatus_t status) {
    assert(status == HIPBLAS_STATUS_SUCCESS);
}

void CallCurand(hiprandStatus status) {
    assert(status == HIPRAND_STATUS_SUCCESS);
}

hipblasHandle_t& GetCublasHandle() {
    static hipblasHandle_t handle;
    static bool init;
    if (!init) {
        init = true;
        CallCublas(hipblasCreate(&handle));
    }
    return handle;
}

NumberPointerArray ToNumberPointerArray(const std::vector<dtype*> &vec) {
    NumberPointerArray device_arr;
    device_arr.init(const_cast<dtype**>(vec.data()), vec.size());
    return device_arr;
}

IntPointerArray ToIntPointerArray(const std::vector<int*> &vec) {
    IntPointerArray device_arr;
    device_arr.init(const_cast<int**>(vec.data()), vec.size());
    return device_arr;
}

IntArray ToIntArray(const std::vector<int> vec) {
    IntArray device_arr;
    device_arr.init(const_cast<int*>(vec.data()), vec.size());
    return device_arr;
}

void NumberPointerArray::init(dtype **host_arr, int len) {
    CallCuda(MemoryPool::Ins().Malloc((void**)&value, len * sizeof(dtype*)));
    CallCuda(hipMemcpy(value, host_arr, len * sizeof(dtype*),
                hipMemcpyHostToDevice));
    this->len = len;
}

NumberPointerArray::~NumberPointerArray() {
    assert(value != NULL);
    CallCuda(MemoryPool::Ins().Free(value));
}

void NumberArray::init(dtype *host_arr, int len) {
    CallCuda(MemoryPool::Ins().Malloc((void**)&value, len * sizeof(dtype)));
    CallCuda(hipMemcpy(value, host_arr, len * sizeof(dtype),
                hipMemcpyHostToDevice));
    this->len = len;
}

NumberArray::~NumberArray() {
    assert(value != NULL);
    CallCuda(MemoryPool::Ins().Free(value));
}

void IntPointerArray::init(int **host_arr, int len) {
    CallCuda(MemoryPool::Ins().Malloc((void**)&value, len * sizeof(int*)));
    CallCuda(hipMemcpy(value, host_arr, len * sizeof(int*),
                hipMemcpyHostToDevice));
    this->len = len;
}

IntPointerArray::~IntPointerArray() {
    assert(value != NULL);
    CallCuda(MemoryPool::Ins().Free(value));
}

void IntArray::init(int *host_arr, int len) {
    CallCuda(MemoryPool::Ins().Malloc((void**)&value, len * sizeof(int)));
    CallCuda(hipMemcpy(value, host_arr, len * sizeof(int),
                hipMemcpyHostToDevice));
    this->len = len;
}

IntArray::~IntArray() {
    assert(value != NULL);
    CallCuda(MemoryPool::Ins().Free(value));
}

void Tensor1D::init(int dim) {
    initOnDevice(dim);
    v = new dtype[dim];
    zero();
}

void Tensor1D::initOnDevice(int dim) {
    CallCuda(MemoryPool::Ins().Malloc((void**)&value, dim * sizeof(dtype)));
    this->dim = dim;
}

Tensor1D::Tensor1D(const Tensor1D &t) {
    dim = t.dim;
    memcpy(v, t.v, dim *sizeof(dtype));
    CallCuda(hipMemcpy(value, t.value, dim * sizeof(dtype),
                hipMemcpyDeviceToDevice));
}

Tensor1D::~Tensor1D() {
    CallCuda(MemoryPool::Ins().Free(value));
    if (v != NULL) {
        delete []v;
    }
}

void Tensor1D::copyFromHostToDevice() {
    assert(v != NULL);
    assert(value != NULL);
    CallCuda(hipMemcpy(value, v, dim * sizeof(dtype), hipMemcpyHostToDevice));
}

void Tensor1D::copyFromDeviceToHost() {
    CallCuda(hipMemcpy(v, value, dim * sizeof(dtype), hipMemcpyDeviceToHost));
}

void Tensor2D::init(int row, int col) {
    initOnDevice(row, col);
    v = new dtype[row * col];
    zero();
}

void Tensor2D::initOnDevice(int row, int col) {
    CallCuda(MemoryPool::Ins().Malloc((void**)&value,
                row * col * sizeof(dtype)));
    this->row = row;
    this->col = col;
    this->size = row * col;
}

Tensor2D::Tensor2D(const Tensor2D &t) {
    row = t.row;
    col = t.col;
    memcpy(v, t.v, sizeof(dtype) * row * col);
    CallCuda(hipMemcpy(value, t.value, sizeof(dtype) * row * col,
                hipMemcpyDeviceToDevice));
}

Tensor2D::~Tensor2D() {
    CallCuda(MemoryPool::Ins().Free(value));
    if (v != NULL) {
        delete [] v;
    }
}

void Tensor2D::copyFromHostToDevice() {
    CallCuda(hipMemcpy(value, v, size * sizeof(dtype), hipMemcpyHostToDevice));
}

void Tensor2D::copyFromDeviceToHost() {
    CallCuda(hipMemcpy(v, value, size * sizeof(dtype), hipMemcpyDeviceToHost));
}

void Random(dtype *v, int len, dtype bound) {
    dtype *mem = (dtype*)malloc(len * sizeof(dtype));
    assert(mem != NULL);
    dtype min = -bound, max = bound;
    for (int i = 0; i < len; i++) {
        mem[i] =  (dtype(rand()) / RAND_MAX) * (max - min) + min;
    }

    CallCuda(hipMemcpy(v, mem, len * sizeof(dtype), hipMemcpyHostToDevice));

    free(mem);
}

__device__ int DeviceDefaultIndex() {
    return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ int DeviceDefaultStep() {
    return gridDim.x * blockDim.x;
}

__device__ dtype DeviceAbs(dtype d) {
    return d > 0 ? d : -d;
}

__global__ void KernelZero(dtype *v, int len) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= len) {
        return;
    }
    v[index] = 0;
}

void Zero(dtype *v, int len) {
    int block_count = (len - 1 + THREAD_COUNT_PER_BLOCK) /
        THREAD_COUNT_PER_BLOCK;
    KernelZero<<<block_count, THREAD_COUNT_PER_BLOCK>>>(v, len);
}

__global__ void PrintPointers(void **p, int len) {
    for (int i = 0; i < len; ++i) {
        printf("%p\n", p[i]);
    }
}

__global__ void PrintNums(dtype* p, int len) {
    for (int i = 0; i < len; ++i) {
        printf("%f\n", p[i]);
    }
}


void InitCuda() {
    hipSetDevice(1);

    cnmemDevice_t device;
    device.size = 10000000000;
    device.device = 1;
    //cnmemInit(1, &device, CNMEM_FLAGS_DEFAULT);

    CallCuda(cudaPrintfInit());
}

void EndCuda() {
    cudaPrintfEnd();
    Profiler::Ins().Print();
}

__global__ void KernelCopyFromOneVectorToMultiVectors(const dtype *src,
        dtype *dest, int count, int len) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < len * count) {
        int count_i = index / len;
        int len_i = index % len;
        dest[count_i * len + len_i] = src[len_i];
    }
}

void CopyFromOneVectorToMultiVectors(const dtype *src, dtype *dest, int count,
        int len) {
    KernelCopyFromOneVectorToMultiVectors<<<
        (len * count - 1 + THREAD_COUNT_PER_BLOCK) / THREAD_COUNT_PER_BLOCK,
    THREAD_COUNT_PER_BLOCK>>>(src, dest, count, len);
}

__global__ void KernelTanh(const dtype *src, dtype**dest, dtype* dest2,
        int count, int len, bool is_being_trained, dtype drop_factor,
        const dtype *drop_mask) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;

    __syncthreads();

    for (int i = index; i < len * count; i += step) {
        int count_i = i % count;
        int len_i = i / count;
        dtype result = cuda_tanh(src[i]);
        if (is_being_trained) {
            if (drop_mask[i] <= drop_factor) {
                dest[count_i][len_i] = 0.0f;
                dest2[i] = result;
            } else {
                dest[count_i][len_i] = result;
                dest2[i] = result;
            }
        } else {
            dest[count_i][len_i] = result * (1 - drop_factor);
            dest2[i] = result;
        }
    }
}

__global__ void KernelCountDrop(dtype *y, int dim) {
    int count = 0;
    for (int i = 0; i < dim; ++i) {
        if (y[i] > -0.0001 && y[i] < 0.0001) {
            ++count;
        }
    }
    KernelPrintLine("drop count:%d", count);
}

void Tanh(const dtype *src, const std::vector<dtype*>& dest, dtype *dest2,
        int len, bool is_being_trained, dtype drop_factor,
        const dtype *drop_mask) {
    if (drop_factor < 0) {
        drop_factor = 0;
    }
    int count = dest.size();
    NumberPointerArray dest_arr = ToNumberPointerArray(dest);
    int block_count = std::min((len * count - 1 + THREAD_COUNT_PER_BLOCK) /
        THREAD_COUNT_PER_BLOCK, BLOCK_COUNT);
    KernelTanh<<<block_count, THREAD_COUNT_PER_BLOCK>>>(src, dest_arr.value,
            dest2, count, len, is_being_trained, drop_factor, drop_mask);
}

__global__ void KernelCopyForUniNodeForward(const dtype** xs, const dtype* b,
        dtype* xs_dest,
        dtype* b_dest,
        int count,
        int x_len,
        int b_len) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    int x_total_len = count * x_len;
    int b_total_len = count * b_len;
    for (int i = index; i < x_total_len + b_total_len; i += step) {
        if (i < x_total_len) {
            int len_i = i / count;
            int count_i = i % count;
            xs_dest[i] = xs[count_i][len_i];
        } else {
            int b_i = i - x_total_len;
            int len_i = b_i / count;
            b_dest[b_i] = b[len_i];
        }
    }
}

void CopyForUniNodeForward(const std::vector<dtype*> &xs, const dtype* b,
        dtype* xs_dest,
        dtype* b_dest,
        int count,
        int x_len,
        int b_len) {
    int len = x_len + b_len;
    int block_count = std::min((count * len - 1 + THREAD_COUNT_PER_BLOCK) / THREAD_COUNT_PER_BLOCK, 56);
    NumberPointerArray xs_arr = ToNumberPointerArray(xs);
    KernelCopyForUniNodeForward<<<block_count, THREAD_COUNT_PER_BLOCK>>>((const dtype**)xs_arr.value,
            (const dtype*)b, xs_dest,
            b_dest,
            count,
            x_len,
            b_len);
}

void MatrixMultiplyMatrix(dtype *W, dtype *x, dtype *y, int row, int col,
        int count, bool useb, bool should_x_transpose,
        bool should_W_transpose) {
    hipblasHandle_t &handle = GetCublasHandle();
    float alpha = 1;
    float beta = useb? 1 : 0;
    hipblasOperation_t x_op = should_x_transpose ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    int ldx = should_x_transpose ? col : count;
    hipblasOperation_t W_op = should_W_transpose ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    int ldw = should_W_transpose ? row : col;
#if USE_FLOAT
    CallCublas(hipblasSgemm(handle, x_op, W_op, count, row, col,
                &alpha, x, ldx, W, ldw, &beta, y, count));
#else
    CallCublas(hipblasDgemm(handle, x_op, W_op, count, row, col,
                &alpha, x, ldx, W, ldw, &beta, y, count));
#endif
}

__global__ void KernelVerify(dtype *host, dtype *device, int len,
        const char *message, bool *success) {
    int index = DeviceDefaultIndex();
    if (index < len) {
        dtype loss = host[index] - device[index];
        if (DeviceAbs(loss) > 0.01) {
            *success = false;
            printf("KernelVerify %s: host:%f device:%f loss:%f\n",
                    message,
                    host[index],
                    device[index],
                    loss);
            KernelPrintLine("KernelVerify: host:%f device:%f loss:%f",
                    host[index],
                    device[index],
                    loss);
        }
    }
}

bool Verify(dtype *host, dtype *device, int len, const char* message) {
    NumberArray arr;
    arr.init(host, len);
    int block_count = (len + THREAD_COUNT_PER_BLOCK - 1) /
        THREAD_COUNT_PER_BLOCK;
    char *m;
    CallCuda(MemoryPool::Ins().Malloc((void**)&m,
                (strlen(message) + 1) * sizeof(char)));
    CallCuda(hipMemcpy(m, message,
                (strlen(message) + 1) * sizeof(char), hipMemcpyHostToDevice));
    bool success = true;
    bool *dev_success;
    CallCuda(MemoryPool::Ins().Malloc((void**)&dev_success, sizeof(bool)));
    CallCuda(hipMemcpy(dev_success, &success, sizeof(bool),
                hipMemcpyHostToDevice));
    KernelVerify<<<block_count, THREAD_COUNT_PER_BLOCK>>>(arr.value, device,
            len, m, dev_success);
    CallCuda(hipMemcpy(&success, dev_success, sizeof(bool),
                hipMemcpyDeviceToHost));
//    if (!success) {
//        printf("host:\n");
//        PrintNums<<<1, 1>>>(arr.value, len);
//        hipDeviceSynchronize();
//        printf("device:\n");
//        PrintNums<<<1, 1>>>(device, len);
//        hipDeviceSynchronize();
//    }
    MemoryPool::Ins().Free(m);
    hipDeviceSynchronize();
    cudaPrintfDisplay(stdout, true);
    return success;
}

hipError_t MemoryPool::Malloc(void **p, int size) {
    //CallCnmem(cnmemMalloc(p, size, NULL));
    //return hipSuccess;

//    return hipMalloc(p, size);

    //std::cout << "free size:" << free_blocks_.size() << " busy size:" <<
    //    busy_blocks_.size() << std::endl;
    Profiler &profiler = Profiler::Ins();
    profiler.BeginEvent("malloc");
    int min_size = 1000000000;
    std::list<MemoryBlock>::iterator min_it = free_blocks_.end();
    for (auto it = free_blocks_.begin(); it != free_blocks_.end(); ++it) {
        if (size <= it->size && min_size > it->size) {
            min_size = it->size;
            min_it = it;
        }
    }

    hipError_t status = hipSuccess;
    if (min_it != free_blocks_.end()) {
        //std::cout << "cache hit" << std::endl;
        busy_blocks_.push_back(*min_it);
        *p = min_it->p;
        free_blocks_.erase(min_it);
    } else {
        //std::cout << "no block, malloc" << std::endl;
        status = hipMalloc(p, size);
        assert(status == hipSuccess);
        MemoryBlock block(*p, size);
        busy_blocks_.push_back(block);
    }

    profiler.EndEvent();
    return status;
}

void MemoryPool::FreePool() {
    if (!busy_blocks_.empty()) {
        std::cout << "warning: busy_blocks_ not empty size:" << busy_blocks_.size() <<std::endl;
        for (MemoryBlock &b : busy_blocks_) {
            CallCuda(hipFree(b.p));
        }
    }

    for (MemoryBlock &b : free_blocks_) {
        CallCuda(hipFree(b.p));
    }
}

hipError_t MemoryPool::Free(void *p) {
    Profiler &profiler = Profiler::Ins();
    profiler.BeginEvent("free");
//    CallCnmem(cnmemFree(p, NULL));

//    return hipFree(p);

    for (auto it = busy_blocks_.end() - 1; it != busy_blocks_.begin() - 1; --it) {
        if (p == it->p) {
            free_blocks_.push_back(*it);
            busy_blocks_.erase(it);
            break;
        }
    }
    profiler.EndEvent();

    return hipSuccess;
}

void Profiler::EndCudaEvent() {
    hipDeviceSynchronize();
    EndEvent();
}

__global__ void KernelCalculateLtyForUniBackward(const dtype *const*ly,
        const dtype *ty,
        const dtype *y,
        const dtype *drop_mask,
        dtype drop_factor,
        dtype *lty,
        int count,
        int dim) {
    int index = DeviceDefaultIndex();
    int step = DeviceDefaultStep();
    int len = count * dim;
    for (int i = index; i < len; i += step) {
        int count_i = i % count;
        int dim_i = i / count;
        dtype yi = y[i];
        if (drop_mask[i] <= drop_factor) {
            lty[i] = 0.0f;
        } else {
            lty[i] = ly[count_i][dim_i] * (1 - yi * yi);
        }
    }
}

void CalculateLtyForUniBackward(const std::vector<dtype*> &ly, const dtype *ty,
        const dtype *y,
        const dtype *drop_mask,
        dtype drop_factor,
        dtype *lty,
        int count,
        int dim) {
    if (drop_factor < 0) {
        drop_factor = 0;
    }
    NumberPointerArray ly_arr = ToNumberPointerArray(ly);
    int block_count = std::min(BLOCK_COUNT, (count * dim +
                THREAD_COUNT_PER_BLOCK - 1) / THREAD_COUNT_PER_BLOCK);
    KernelCalculateLtyForUniBackward<<<block_count,
        THREAD_COUNT_PER_BLOCK>>>(ly_arr.value, ty, y, drop_mask, drop_factor,
                lty, count, dim);
}

__device__ int global_block_count[1000000];
__global__ void KernelAddLtyToParamBiasAndAddLxToInputLossesForUniBackward(
        const dtype *lty,
        const dtype *lx,
        dtype *b,
        dtype **losses,
        int count,
        int out_dim,
        int in_dim,
        dtype *block_sums) {
    __shared__ volatile dtype shared_arr[THREAD_COUNT_PER_BLOCK];

    int count_i = blockIdx.y * blockDim.x + threadIdx.x;
    //if (count >= THREAD_COUNT_PER_BLOCK) {
    //    KernelPrintLine("count_i:%d", count_i);
    //}
    int dim_i = blockIdx.x;
    if (dim_i < out_dim) {
        if (threadIdx.x == 0 && blockIdx.y == 0) {
            global_block_count[dim_i] = 0;
        }
        int lty_index = dim_i * count + count_i;
        shared_arr[threadIdx.x] = count_i < count ? lty[lty_index] : 0.0f;
        __syncthreads();

        for (int i = (THREAD_COUNT_PER_BLOCK >> 1); i > 0; i>>=1) {
            if (threadIdx.x < i) {
                shared_arr[threadIdx.x] += shared_arr[threadIdx.x + i];
            }
            __syncthreads();
        }

        if (threadIdx.x == 0) {
            block_sums[gridDim.y * blockIdx.x + blockIdx.y] = shared_arr[0];
            if (atomicAdd(global_block_count + dim_i, 1) == gridDim.y - 1) {
                dtype sum = 0.0;
                for (int i = 0; i < gridDim.y; ++i) {
                    sum += block_sums[gridDim.y * blockIdx.x + i];
                }
                b[dim_i] += sum;
            }
        }
    } else {
        if (count_i < count) {
            dim_i -= out_dim;
            int lx_index = dim_i * count + count_i;
            losses[count_i][dim_i] += lx[lx_index];
        }
    }
}

void AddLtyToParamBiasAndAddLxToInputLossesForUniBackward(const dtype *lty,
        const dtype *lx, dtype *b, std::vector<dtype*> &losses, int count,
        int out_dim, int in_dim) {
    int block_y = (count - 1 + THREAD_COUNT_PER_BLOCK) /
        THREAD_COUNT_PER_BLOCK;
    dim3 block_dim(out_dim + in_dim, block_y, 1);
    NumberPointerArray loss_arr;
    loss_arr.init(losses.data(), count);
    Tensor1D block_sums;
    block_sums.init(block_y * out_dim);
    KernelAddLtyToParamBiasAndAddLxToInputLossesForUniBackward<<<block_dim,
        THREAD_COUNT_PER_BLOCK>>>(lty, lx, b, loss_arr.value, count, out_dim,
                in_dim, block_sums.value);
    //cudaPrintfDisplay(stdout, true);
}

constexpr int MAX_BATCH_COUNT = 1000000;

__global__ void KernelInitCurandStates(hiprandState_t *states) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    for (int i = index; i < MAX_BATCH_COUNT; i += step) {
        hiprand_init(0, i, 0, &states[i]);
    }
}

hiprandState_t *GetCurandStates() {
    static hiprandState_t *states;
    if (states == NULL) {
        MemoryPool &pool = MemoryPool::Ins();
        CallCuda(pool.Malloc((void**)&states, sizeof(hiprandState_t) *
                    MAX_BATCH_COUNT));
        KernelInitCurandStates<<<BLOCK_COUNT, THREAD_COUNT_PER_BLOCK>>>(
                states);
    }
    return states;
}

hiprandGenerator_t &GetGenerator() {
    static hiprandGenerator_t gen;
    static bool init;
    if (!init) {
        CallCurand(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
        CallCurand(hiprandSetPseudoRandomGeneratorSeed(gen, 0));
        init = true;
    }
    return gen;
}

void CalculateDropoutMask(dtype drop_factor, int count, int dim, dtype* mask) {
    hiprandGenerator_t &gen = GetGenerator();
    CallCurand(hiprandGenerateUniform(gen, mask, count * dim));
}

}
