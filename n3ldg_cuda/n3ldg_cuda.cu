#include "hip/hip_runtime.h"
#include "n3ldg_cuda.h"
#include <cstdlib>
#include <vector>
#include <algorithm>
#include <cmath>
#include <cstdio>
#include <hipblas.h>
#include "cuPrintf.cuh"
#include "cuPrintf.cu"
#include "memory_pool.h"
#include "profiler.h"
#include "cnmem.h"

namespace n3ldg_cuda {

using std::cout;
using std::endl;

#define cuda_sqrt(x) sqrtf(x)
#define cuda_pow(x, y) powf(x, y)
#define cuda_tanh(x) tanhf(x)

#define KERNEL_LOG

#ifdef KERNEL_LOG
#define  KernelPrintLine(format, ...)\
{\
    cuPrintf("block:x=%d,y=%d thread:x=%d,y=%d "#format"\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y,\
            __VA_ARGS__);\
}
#else
#define KernelPrintLine(format, ...)
#endif

constexpr int THREAD_COUNT_PER_BLOCK = 1024;
constexpr int BLOCK_COUNT = 56;

void CallCuda(hipError_t status) {
    if (status != hipSuccess) {
        cout << hipGetErrorString(status) << endl;
        abort();
    }
}

void CallCnmem(cnmemStatus_t status) {
    assert(status == CNMEM_STATUS_SUCCESS);
}

void CallCublas(hipblasStatus_t status) {
    assert(status == HIPBLAS_STATUS_SUCCESS);
}

hipblasHandle_t& GetCublasHandle() {
    static hipblasHandle_t handle;
    static bool init;
    if (!init) {
        init = true;
        CallCublas(hipblasCreate(&handle));
    }
    return handle;
}

NumberPointerArray ToNumberPointerArray(const std::vector<dtype*> &vec) {
    NumberPointerArray device_arr;
    device_arr.init(const_cast<dtype**>(vec.data()), vec.size());
    return device_arr;
}

IntPointerArray ToIntPointerArray(const std::vector<int*> &vec) {
    IntPointerArray device_arr;
    device_arr.init(const_cast<int**>(vec.data()), vec.size());
    return device_arr;
}

IntArray ToIntArray(const std::vector<int> vec) {
    IntArray device_arr;
    device_arr.init(const_cast<int*>(vec.data()), vec.size());
    return device_arr;
}

void NumberPointerArray::init(dtype **host_arr, int len) {
    CallCuda(MemoryPool::Ins().Malloc((void**)&value, len * sizeof(dtype*)));
    CallCuda(hipMemcpy(value, host_arr, len * sizeof(dtype*),
                hipMemcpyHostToDevice));
    this->len = len;
}

NumberPointerArray::~NumberPointerArray() {
    assert(value != NULL);
    CallCuda(MemoryPool::Ins().Free(value));
}

void NumberArray::init(dtype *host_arr, int len) {
    CallCuda(MemoryPool::Ins().Malloc((void**)&value, len * sizeof(dtype)));
    CallCuda(hipMemcpy(value, host_arr, len * sizeof(dtype),
                hipMemcpyHostToDevice));
    this->len = len;
}

NumberArray::~NumberArray() {
    assert(value != NULL);
    CallCuda(MemoryPool::Ins().Free(value));
}

void IntPointerArray::init(int **host_arr, int len) {
    CallCuda(MemoryPool::Ins().Malloc((void**)&value, len * sizeof(int*)));
    CallCuda(hipMemcpy(value, host_arr, len * sizeof(int*),
                hipMemcpyHostToDevice));
    this->len = len;
}

IntPointerArray::~IntPointerArray() {
    assert(value != NULL);
    CallCuda(MemoryPool::Ins().Free(value));
}

void IntArray::init(int *host_arr, int len) {
    CallCuda(MemoryPool::Ins().Malloc((void**)&value, len * sizeof(int)));
    CallCuda(hipMemcpy(value, host_arr, len * sizeof(int),
                hipMemcpyHostToDevice));
    this->len = len;
}

IntArray::~IntArray() {
    assert(value != NULL);
    CallCuda(MemoryPool::Ins().Free(value));
}

void Tensor1D::init(int dim) {
    CallCuda(MemoryPool::Ins().Malloc((void**)&value, dim * sizeof(dtype)));
    this->dim = dim;
    v = new dtype[dim];
    zero();
}

Tensor1D::Tensor1D(const Tensor1D &t) {
    dim = t.dim;
    memcpy(v, t.v, dim *sizeof(dtype));
    CallCuda(hipMemcpy(value, t.value, dim * sizeof(dtype),
                hipMemcpyDeviceToDevice));
}

Tensor1D::~Tensor1D() {
    assert(value != NULL && v != NULL);
    CallCuda(MemoryPool::Ins().Free(value));
    delete []v;
}

void Tensor1D::copyFromHostToDevice() {
    assert(v != NULL);
    assert(value != NULL);
    CallCuda(hipMemcpy(value, v, dim * sizeof(dtype), hipMemcpyHostToDevice));
}

void Tensor1D::copyFromDeviceToHost() {
    CallCuda(hipMemcpy(v, value, dim * sizeof(dtype), hipMemcpyDeviceToHost));
}

void Tensor2D::init(int row, int col) {
    CallCuda(MemoryPool::Ins().Malloc((void**)&value, row * col * sizeof(dtype)));
    v = new dtype[row * col];
    this->row = row;
    this->col = col;
    this->size = row * col;
    zero();
}

Tensor2D::Tensor2D(const Tensor2D &t) {
    row = t.row;
    col = t.col;
    memcpy(v, t.v, sizeof(dtype) * row * col);
    CallCuda(hipMemcpy(value, t.value, sizeof(dtype) * row * col,
                hipMemcpyDeviceToDevice));
}

Tensor2D::~Tensor2D() {
    assert(value != NULL && v != NULL);
    CallCuda(MemoryPool::Ins().Free(value));
    delete [] v;
}

void Tensor2D::copyFromHostToDevice() {
    CallCuda(hipMemcpy(value, v, size * sizeof(dtype), hipMemcpyHostToDevice));
}

void Tensor2D::copyFromDeviceToHost() {
    CallCuda(hipMemcpy(v, value, size * sizeof(dtype), hipMemcpyDeviceToHost));
}

__global__ void KernelUpdateAdam(dtype *val,  dtype *grad,
        dtype *aux_mean, dtype *aux_square, int row, int col, int iter, dtype belta1,
        dtype belta2, dtype alpha, dtype reg, dtype eps) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int len = row * col;
    if (index >= len) {
        return;
    }
    if (col > 1 && row > 1) {
        grad[index] = grad[index] + reg * val[index];
    }
    __syncthreads();

    aux_mean[index] = belta1 * aux_mean[index] + (1-belta1) * grad[index];
    aux_square[index] = belta2 * aux_square[index] + (1 - belta2) *
        grad[index] * grad[index];

    dtype lr_t = alpha * cuda_sqrt(1 - cuda_pow(belta2, iter + 1)) /
        (1 - cuda_pow(belta1, iter + 1));
    val[index] = val[index] - aux_mean[index] * lr_t /
        cuda_sqrt(aux_square[index] + eps);
}

void UpdateAdam(Tensor2D &val, Tensor2D &grad, Tensor2D &aux_mean,
        Tensor2D &aux_square, int &iter, dtype belta1, dtype belta2,
        dtype alpha, dtype reg, dtype eps) {
    int block_count = (val.row * val.col - 1 + THREAD_COUNT_PER_BLOCK) /
        THREAD_COUNT_PER_BLOCK;
    KernelUpdateAdam<<<block_count, THREAD_COUNT_PER_BLOCK>>>(val.value,
            grad.value, aux_mean.value, aux_square.value, val.row, val.col,
            iter, belta1, belta2, alpha, reg, eps);
    ++iter;
}

__device__ volatile dtype global_sum_temp[100][THREAD_COUNT_PER_BLOCK];
__device__ int block_nums_in_x[100];
__device__ int block_num_in_y;
__device__ volatile dtype global_sum;

__global__ void KernelRescaleGrads(dtype **grads, int *lens,
        dtype max_scale) {
    __shared__ volatile dtype sum_temp[THREAD_COUNT_PER_BLOCK];
    __shared__ volatile bool is_last_block_in_x;

    if (threadIdx.x == 0) {
        is_last_block_in_x = false;
    }

    dtype *grad = grads[blockIdx.y];
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index == 0) {
        block_nums_in_x[blockIdx.y] = 0;
        if (blockIdx.y == 0) {
            block_num_in_y = 0;
        }
    }
    int len = lens[blockIdx.y];
    if (index < len) {
        dtype grad_val = grad[index];
        sum_temp[threadIdx.x] = grad_val * grad_val;
    } else {
        sum_temp[threadIdx.x] = 0;
    }
    __syncthreads();

    for (int i = (THREAD_COUNT_PER_BLOCK >> 1); i > 0; i >>=1) {
        if (threadIdx.x < i) {
            sum_temp[threadIdx.x] += sum_temp[threadIdx.x + i];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        global_sum_temp[blockIdx.y][blockIdx.x] = sum_temp[0];
        if (atomicAdd(block_nums_in_x + blockIdx.y, 1) == gridDim.x - 1) {
            is_last_block_in_x = true;
        }
    }
    __syncthreads();

    if (is_last_block_in_x) {
        if (threadIdx.x < gridDim.x) {
            sum_temp[threadIdx.x] = global_sum_temp[blockIdx.y][threadIdx.x];
        } else {
            sum_temp[threadIdx.x] = 0;
        }
        __syncthreads();

        for (int i = (THREAD_COUNT_PER_BLOCK >> 1); i > 0; i >>=1) {
            if (threadIdx.x < i) {
                sum_temp[threadIdx.x] += sum_temp[threadIdx.x + i];
            }
            __syncthreads();
        }

        if (threadIdx.x == 0) {
            global_sum_temp[0][blockIdx.y] = sum_temp[0];
            if (atomicAdd(&block_num_in_y, 1) == gridDim.y - 1) {
                dtype sum = 0;
                for (int i = 0; i<gridDim.y; ++i) {
                    sum += global_sum_temp[0][i];
                }
                global_sum = sum;
            }
        }
    }
    __syncthreads();
    int local_global_sum = global_sum;

    assert(local_global_sum < 1e20);
    dtype norm = cuda_sqrt(local_global_sum);
    if (max_scale > 0 && norm > max_scale) {
        dtype scale = max_scale / norm;
        grad[index] *= scale;
    }
}

void RescaleGrads(std::vector<dtype *> &grads, const std::vector<int> &lens,
        dtype max_scale) {
    assert(grads.size() == lens.size());
    assert(grads.size() <= 100);
    int max_len = *std::max_element(lens.begin(), lens.end());
    std::cout << "max_len:" << max_len << std::endl;
    assert(max_len < THREAD_COUNT_PER_BLOCK * THREAD_COUNT_PER_BLOCK);

    int block_count = (max_len - 1 + THREAD_COUNT_PER_BLOCK) /
        THREAD_COUNT_PER_BLOCK;
    dim3 block_dim(block_count, grads.size(), 1);

    NumberPointerArray grads_arr = ToNumberPointerArray(grads);
    IntArray len_arr = ToIntArray(lens);

    KernelRescaleGrads<<<block_dim, THREAD_COUNT_PER_BLOCK>>>(grads_arr.value,
            len_arr.value, max_scale);
}

void Random(dtype *v, int len, dtype bound) {
    dtype *mem = (dtype*)malloc(len * sizeof(dtype));
    assert(mem != NULL);
    dtype min = -bound, max = bound;
    for (int i = 0; i < len; i++) {
        mem[i] =  (dtype(rand()) / RAND_MAX) * (max - min) + min;
    }

    CallCuda(hipMemcpy(v, mem, len * sizeof(dtype), hipMemcpyHostToDevice));

    free(mem);
}

__global__ void KernelZero(dtype *v, int len) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= len) {
        return;
    }
    v[index] = 0;
}

void Zero(dtype *v, int len) {
    int block_count = (len - 1 + THREAD_COUNT_PER_BLOCK) /
        THREAD_COUNT_PER_BLOCK;
    KernelZero<<<block_count, THREAD_COUNT_PER_BLOCK>>>(v, len);
}

__global__ void PrintPointers(void **p, int len) {
    for (int i = 0; i < len; ++i) {
        printf("%p\n", p[i]);
    }
}

__global__ void PrintNums(dtype* p, int len) {
    for (int i = 0; i < len; ++i) {
        printf("%f,", p[i]);
    }
    printf("\n");
}


void InitCuda() {
    //hipSetDevice(1);

    cnmemDevice_t device;
    device.size = 10000000000;
    device.device = 1;
    cnmemInit(1, &device, CNMEM_FLAGS_DEFAULT);

    CallCuda(cudaPrintfInit());
}

void EndCuda() {
    cudaPrintfEnd();
    Profiler::Ins().Print();
}

__global__ void KernelCopyFromOneVectorToMultiVectors(const dtype *src,
        dtype *dest, int count, int len) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < len * count) {
        int count_i = index / len;
        int len_i = index % len;
        dest[count_i * len + len_i] = src[len_i];
    }
}

void CopyFromOneVectorToMultiVectors(const dtype *src, dtype *dest, int count, int len) {
    KernelCopyFromOneVectorToMultiVectors<<<
        (len * count - 1 + THREAD_COUNT_PER_BLOCK) / THREAD_COUNT_PER_BLOCK, THREAD_COUNT_PER_BLOCK>>>(
                src, dest, count, len);
}

__global__ void Tanh(const dtype *src, dtype**dest, dtype* dest2, int count, int len) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    for (int i = index; i < len * count; i += step) {
        int count_i = i % count;
        int len_i = i / count;
        dtype result = cuda_tanh(src[i]);
        dest[count_i][len_i] = result;
        dest2[i] = result;
    }
}

void Tanh(const dtype *src, const std::vector<dtype*>& dest, dtype *dest2, int len) {
    int count = dest.size();
    NumberPointerArray dest_arr = ToNumberPointerArray(dest);
    int block_count = std::min((len * count - 1 + THREAD_COUNT_PER_BLOCK) /
        THREAD_COUNT_PER_BLOCK, BLOCK_COUNT);
    Tanh<<<block_count, THREAD_COUNT_PER_BLOCK>>>(src, dest_arr.value, dest2, count, len);
}

__global__ void KernelCopyForUniNodeForward(const dtype** xs, const dtype* b,
        dtype* xs_dest,
        dtype* b_dest,
        int count,
        int x_len,
        int b_len) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    int x_total_len = count * x_len;
    for (int i = index; i < x_total_len; i += step) {
        int len_i = i / count;
        int count_i = i % count;
        xs_dest[i] = xs[count_i][len_i];
    }
    for (int i = index; i < x_total_len + count * b_len; i += step) {
        int b_i = i - x_total_len;
        int len_i = b_i / count;
        b_dest[b_i] = b[len_i];
    }
}

void CopyForUniNodeForward(const std::vector<dtype*> &xs, const dtype* b,
        dtype* xs_dest,
        dtype* b_dest,
        int count,
        int x_len,
        int b_len) {
    int len = x_len + b_len;
    int block_count = std::min((count * len - 1 + THREAD_COUNT_PER_BLOCK) / THREAD_COUNT_PER_BLOCK, 56);
    NumberPointerArray xs_arr = ToNumberPointerArray(xs);
    KernelCopyForUniNodeForward<<<block_count, THREAD_COUNT_PER_BLOCK>>>((const dtype**)xs_arr.value,
            (const dtype*)b, xs_dest,
            b_dest,
            count,
            x_len,
            b_len);
}

void MatrixMultiplyMatrix(dtype *W, dtype *x, dtype *y, int row, int col, int count, bool useb) {
    hipblasHandle_t &handle = GetCublasHandle();
    float alpha = 1;
    float beta = useb? 1 : 0;
#if USE_FLOAT
    CallCublas(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, count, row, col, &alpha, x, count, W, col, &beta,
            y,
            count));
#else
    CallCublas(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, count, row, col, &alpha, x, count, W, col, &beta,
            y,
            count));
#endif
}

__global__ void KernelVerify(dtype *host, dtype *device, int len) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < len) {
        dtype loss = host[index] - device[index];
        if (loss > 0.01 || loss < -0.01) {
            KernelPrintLine("KernelVerify: host:%f device:%f loss:%f",
                    host[index],
                    device[index],
                    loss);
        }
    }
}

void Verify(dtype *host, dtype *device, int len) {
    NumberArray arr;
    arr.init(host, len);
    int block_count = (len + THREAD_COUNT_PER_BLOCK - 1) /
        THREAD_COUNT_PER_BLOCK;
    KernelVerify<<<block_count, THREAD_COUNT_PER_BLOCK>>>(arr.value, device,
            len);
    hipDeviceSynchronize();
    cudaPrintfDisplay(stdout, true);
}

hipError_t MemoryPool::Malloc(void **p, int size) {
    CallCnmem(cnmemMalloc(p, size, NULL));
    return hipSuccess;

    //return hipMalloc(p, size);

//    bool found = false;
//    for (auto it = free_blocks_.begin(); it != free_blocks_.end(); ++it) {
//        if (size == it->size) {
//            busy_blocks_.push_back(*it);
//            *p = it->p;
//            free_blocks_.erase(it);
//            found = true;
//            break;
//        }
//    }
//
//    hipError_t status = hipSuccess;
//    if (!found) {
//        status = hipMalloc(p, size);
//        assert(status == hipSuccess);
//        MemoryBlock block(*p, size);
//        busy_blocks_.push_back(block);
//    }
//
//    return status;
}

void MemoryPool::FreePool() {
    if (!busy_blocks_.empty()) {
        std::cout << "warning: busy_blocks_ not empty size:" << busy_blocks_.size() <<std::endl;
        for (MemoryBlock &b : busy_blocks_) {
            CallCuda(hipFree(b.p));
        }
    }

    for (MemoryBlock &b : free_blocks_) {
        CallCuda(hipFree(b.p));
    }
}

hipError_t MemoryPool::Free(void *p) {
    CallCnmem(cnmemFree(p, NULL));

//    return hipFree(p);

//    for (auto it = busy_blocks_.begin(); it != busy_blocks_.end(); ++it) {
//        if (p == it->p) {
//            free_blocks_.push_back(*it);
//            busy_blocks_.erase(it);
//            break;
//        }
//    }

    return hipSuccess;
}

void Profiler::EndCudaEvent() {
    hipDeviceSynchronize();
    EndEvent();
}

__global__ void KernelLtyForUniBackward(const dtype **ly, const dtype *ty,
        const dtype *y,
        dtype *lty,
        int count,
        int dim) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int step = blockDim.x * gridDim.x;
    int len = count * dim;
    for (int i = index; i < len; i += step) {
        int count_i = i % count;
        int dim_i = i / count;
        dtype tyi = ty[i];
        lty[i] = ly[count_i][dim_i] * (1 - tyi * tyi);
    }
}

void LtyForUniBackward(const std::vector<dtype*> &ly, const dtype *ty,
        const dtype *y,
        dtype *lty,
        int count,
        int dim) {
    int block_count = std::min(BLOCK_COUNT,
            (count * dim + THREAD_COUNT_PER_BLOCK - 1) /
            THREAD_COUNT_PER_BLOCK);
    NumberPointerArray ly_arr = ToNumberPointerArray(ly);
    KernelLtyForUniBackward<<<block_count, THREAD_COUNT_PER_BLOCK>>>(ly_arr,
            ty, y, lty, count, dim);
}

}
